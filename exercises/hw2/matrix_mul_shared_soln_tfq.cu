
#include <hip/hip_runtime.h>
#include <stdio.h>

// these are just for timing measurments
#include <time.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const int DSIZE = 8192;
const int block_size = 32;  // CUDA maximum is 1024 *total* threads in block
const float A_val = 3.0f;
const float B_val = 2.0f;

// matrix multiply (naive) kernel: C = A * B
__global__ void mmul(const float *A, const float *B, float *C, int ds) {

  int idx = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
  int idy = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

  if ((idx < ds) && (idy < ds)){
    float temp = 0;
    for (int i = 0; i < ds; i++)
      temp += A[idy*ds+i] * B[i*ds+idx];   // dot product of row and column
    C[idy*ds+idx] = temp;
  }
}

// matrix multiply (naive) kernel: C = A * B
// using shared memory
__global__ void mmul_sm(const float *A, const float *B, float *C, int ds) {

  // declare cache in shared memory
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];

  int idx = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
  int idy = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

  if ((idx < ds) && (idy < ds)){
    float temp = 0;
    for (int i = 0; i < ds/block_size; i++) {

      // Load data into shared memory
      As[threadIdx.y][threadIdx.x] = A[idy * ds + (i * block_size + threadIdx.x)];
      Bs[threadIdx.y][threadIdx.x] = B[(i * block_size + threadIdx.y) * ds + idx];

      // Synchronize
      __syncthreads();

      // Keep track of the running sum
      for (int k = 0; k < block_size; k++)
      	temp += As[threadIdx.y][k] * Bs[k][threadIdx.x]; // dot product of row and column
      __syncthreads();

    }

    // Write to global memory
    C[idy*ds+idx] = temp;
  }
}

int main(){

  float *h_A, *h_B, *h_C, *h_C_sm, *d_A, *d_B, *d_C;

  dim3 block(block_size, block_size);  // dim3 variable holds 3 dimensions
  dim3 grid((DSIZE+block.x-1)/block.x, (DSIZE+block.y-1)/block.y);

  // these are just for timing
  clock_t t0, t1, t2, t3;
  double t1sum=0.0;
  double t2sum=0.0;
  double t3sum=0.0;

  // start timing
  t0 = clock();

  h_A = new float[DSIZE*DSIZE];
  h_B = new float[DSIZE*DSIZE];
  h_C = new float[DSIZE*DSIZE];
  h_C_sm = new float[DSIZE*DSIZE];
  for (int i = 0; i < DSIZE*DSIZE; i++){
    h_A[i] = (float) (rand() % 10); // random value between 0 and 9 inclusive
    h_B[i] = (float) (rand() % 10); // random value between 0 and 9 inclusive
    h_C[i] = 0;
    h_C_sm[i] = 0;
  }

  // Initialization timing
  t1 = clock();
  t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
  printf("Init took %f seconds.  Begin compute\n", t1sum);


  // Start non shared memory multiplication //

  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  // Cuda processing sequence step 1 is complete

  // Launch kernel
  mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");

  // Cuda processing sequence step 2 is complete

  // Copy results back to host
  hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

  // GPU timing
  t2 = clock();
  t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
  printf ("Done without shared memory. Compute took %f seconds\n", t2sum);

  // Cuda processing sequence step 3 is complete
  // Free all memory in allocated in GPU
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
    
  // Start shared memory multiplication //

  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  // Cuda processing sequence step 1 is complete

  // Launch kernel
  mmul_sm<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");

  // Cuda processing sequence step 2 is complete

  // Copy results back to host
  hipMemcpy(h_C_sm, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

  // Cuda processing sequence step 3 is complete
  // Free all memory in allocated in GPU
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // GPU timing
  t3 = clock();
  t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
  printf ("Done with shared memory. Compute took %f seconds\n", t3sum);

  // Verify results
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  for (int i = 0; i < DSIZE*DSIZE; i++) if (h_C[i] != h_C_sm[i]) {printf("mismatch at index %d, was: %f, should be: %f\n", i, h_C_sm[i], h_C[i]); return -1;}
  printf("Success!\n"); 
  return 0;
}
  
